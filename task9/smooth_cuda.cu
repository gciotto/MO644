/*
 * Task #9 - Parallel Programming
 * 
 * Gustavo Ciotto Pinton
 */

#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>
#include <math.h>

#define COMMENT "Histogram_GPU"
#define RGB_COMPONENT_COLOR 255

#include <hip/hip_runtime.h>

#define THREAD_PER_BLOCK 1024 /* Tesla k40 supports 1024 threads per block */

/* Mask attributes  */
#define MASK_WIDTH 5
#define RADIUS (MASK_WIDTH-1)/2

#define PIXEL(R,G,B) ( (PPMPixel) { .red = (R), .green = (G), .blue = (B)})

typedef struct {
    unsigned char red, green, blue;
} PPMPixel;

typedef struct {
    int x, y;
    PPMPixel *data;
} PPMImage;

double rtclock()
{
    struct timezone Tzp;
    struct timeval Tp;
    int stat;
    stat = gettimeofday (&Tp, &Tzp);
    if (stat != 0) printf("Error return from gettimeofday: %d",stat);
    return(Tp.tv_sec + Tp.tv_usec*1.0e-6);
}


static PPMImage *readPPM(const char *filename) {
    char buff[16];
    PPMImage *img;
    FILE *fp;
    int c, rgb_comp_color;
    fp = fopen(filename, "rb");
    if (!fp) {
        fprintf(stderr, "Unable to open file '%s'\n", filename);
        exit(1);
    }

    if (!fgets(buff, sizeof(buff), fp)) {
        perror(filename);
        exit(1);
    }

    if (buff[0] != 'P' || buff[1] != '6') {
        fprintf(stderr, "Invalid image format (must be 'P6')\n");
        exit(1);
    }

    img = (PPMImage *) malloc(sizeof(PPMImage));
    if (!img) {
        fprintf(stderr, "Unable to allocate memory\n");
        exit(1);
    }

    c = getc(fp);
    while (c == '#') {
        while (getc(fp) != '\n')
            ;
        c = getc(fp);
    }

    ungetc(c, fp);
    if (fscanf(fp, "%d %d", &img->x, &img->y) != 2) {
        fprintf(stderr, "Invalid image size (error loading '%s')\n", filename);
        exit(1);
    }

    if (fscanf(fp, "%d", &rgb_comp_color) != 1) {
        fprintf(stderr, "Invalid rgb component (error loading '%s')\n",
                filename);
        exit(1);
    }

    if (rgb_comp_color != RGB_COMPONENT_COLOR) {
        fprintf(stderr, "'%s' does not have 8-bits components\n", filename);
        exit(1);
    }

    while (fgetc(fp) != '\n')
        ;
    img->data = (PPMPixel*) malloc(img->x * img->y * sizeof(PPMPixel));

    if (!img) {
        fprintf(stderr, "Unable to allocate memory\n");
        exit(1);
    }

    if (fread(img->data, 3 * img->x, img->y, fp) != img->y) {
        fprintf(stderr, "Error loading image '%s'\n", filename);
        exit(1);
    }

    fclose(fp);
    return img;
}

void writePPM(PPMImage *img) {

    fprintf(stdout, "P6\n");
    fprintf(stdout, "# %s\n", COMMENT);
    fprintf(stdout, "%d %d\n", img->x, img->y);
    fprintf(stdout, "%d\n", RGB_COMPONENT_COLOR);

    fwrite(img->data, 3 * img->x, img->y, stdout);
    fclose(stdout);
}

/* kernel function. It loads MASK_WIDTH rows containing the same number of elements as the number of threads per block +2 
 into the shared memory and computes the new value for the pixel based on this data. */
__global__ void cudaSmoothing (PPMPixel *data_in, PPMPixel *data_out, int columns, int rows) {

	__shared__ PPMPixel shared_data [MASK_WIDTH][THREAD_PER_BLOCK + 2 * RADIUS];

	int i = threadIdx.x + blockIdx.x * blockDim.x,
		shared_i = threadIdx.x + RADIUS,
		n = rows * columns, j, k,
		total_red, total_blue, total_green;
	

	if ( i  < n ) {

		/* Populating shared memory */
		for (j = 0; j < MASK_WIDTH; j++) {

			/* Computes the index of the array corresponding to the column in row (j - RADIUS) */
			int col_index = i + (j - RADIUS) * columns;

			shared_data [j][shared_i] = PIXEL(0,0,0);

			/* only if i is inside the image border */
			if (col_index >= 0 && col_index < n)
				shared_data [j][shared_i] = data_in [col_index];

			if (threadIdx.x < RADIUS) {

				shared_data [j][shared_i - RADIUS] = PIXEL(0,0,0);
				shared_data [j][shared_i + THREAD_PER_BLOCK] = PIXEL(0,0,0);

				if ((col_index - RADIUS) >= 0)
					shared_data [j][shared_i - RADIUS] = data_in [col_index - RADIUS];

				if ((col_index + THREAD_PER_BLOCK) < n) 
					shared_data [j][shared_i + THREAD_PER_BLOCK] = data_in [col_index + THREAD_PER_BLOCK];
			}

		}

		/* Ensures all threads updated the shared memory */
		__syncthreads();

		total_red = total_blue = total_green = 0;

		int left_border = i - (i % columns),
			right_border = i + columns - (i % columns);

		/* Iterates over lines */
		for (j = 0; j < MASK_WIDTH; j++) {
			
			/* Iterates over columns */
			for (k = - RADIUS; k <= RADIUS; k++) {

				/* We must check if the current element is not in the border. In this case, we need to avoid
				   summing the other side element */
				if (i + k >= left_border && i + k < right_border) {

					total_red += shared_data[j][shared_i +  k].red;
					total_blue += shared_data[j][shared_i + k].blue;
					total_green += shared_data[j][shared_i + k].green;
				}
			}

		}

		data_out[i].red = total_red / ( MASK_WIDTH * MASK_WIDTH );
		data_out[i].blue = total_blue / ( MASK_WIDTH * MASK_WIDTH );
		data_out[i].green = total_green / ( MASK_WIDTH * MASK_WIDTH );
	}

}

int main(int argc, char *argv[]) {

    if( argc != 2 ) {
        printf("Too many or no one arguments supplied.\n");
    }

#ifdef PRINT_TIME
    double t_start, t_end;
#endif

    char *filename = argv[1]; //Recebendo o arquivo!;
    PPMImage *image = readPPM(filename);
    PPMImage *image_output = readPPM(filename);

    /* Number of elements in the image */
    int n = image->x * image->y;

#ifdef PRINT_TIME
    t_start = rtclock();
#endif

    /* Allocating memory for image data in the device */
    int image_size = n * sizeof(PPMPixel);
    PPMPixel *cuda_image_data;
    hipMalloc((void**) &cuda_image_data, image_size);

    /* Copying image data to the device */
    hipMemcpy(cuda_image_data, image->data, image_size, hipMemcpyHostToDevice);

    /* Allocating memory for image result in the device */
    PPMPixel *cuda_image_out;
    hipMalloc((void**) &cuda_image_out, image_size);

    /* Computes how many blocks will be used. */
    int cuda_blocks = ceil ( (float) n / THREAD_PER_BLOCK );

    cudaSmoothing <<< cuda_blocks, THREAD_PER_BLOCK >>> (cuda_image_data, cuda_image_out, image->x, image->y);

    /* Copying computed result from device memory */
    hipMemcpy(image_output->data, cuda_image_out, image_size, hipMemcpyDeviceToHost);

#ifdef PRINT_TIME
    t_end = rtclock();
	fprintf(stdout, "\n%0.6lfs\n", t_end - t_start);
#else
    writePPM(image_output);
#endif

    free(image->data);
    free(image);
    free(image_output->data);
    free(image_output);

    hipFree(cuda_image_data);
    hipFree(cuda_image_out);
}

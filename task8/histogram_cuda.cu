/*
  Task #8 - Gustavo Ciotto Pinton
  MO644 - Parallel Programming
*/
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>
#include <math.h>

#define COMMENT "Histogram_GPU"
#define RGB_COMPONENT_COLOR 255

#include <hip/hip_runtime.h>

#define THREAD_PER_BLOCK 1024 /* Tesla k40 supports 1024 threads per block */

typedef struct {
	unsigned char red, green, blue;
} PPMPixel;

typedef struct {
	int x, y;
	PPMPixel *data;
} PPMImage;

double rtclock()
{
    struct timezone Tzp;
    struct timeval Tp;
    int stat;
    stat = gettimeofday (&Tp, &Tzp);
    if (stat != 0) printf("Error return from gettimeofday: %d",stat);
    return(Tp.tv_sec + Tp.tv_usec*1.0e-6);
}


static PPMImage *readPPM(const char *filename) {
	char buff[16];
	PPMImage *img;
	FILE *fp;
	int c, rgb_comp_color;
	fp = fopen(filename, "rb");
	if (!fp) {
		fprintf(stderr, "Unable to open file '%s'\n", filename);
		exit(1);
	}

	if (!fgets(buff, sizeof(buff), fp)) {
		perror(filename);
		exit(1);
	}

	if (buff[0] != 'P' || buff[1] != '6') {
		fprintf(stderr, "Invalid image format (must be 'P6')\n");
		exit(1);
	}

	img = (PPMImage *) malloc(sizeof(PPMImage));
	if (!img) {
		fprintf(stderr, "Unable to allocate memory\n");
		exit(1);
	}

	c = getc(fp);
	while (c == '#') {
		while (getc(fp) != '\n')
			;
		c = getc(fp);
	}

	ungetc(c, fp);
	if (fscanf(fp, "%d %d", &img->x, &img->y) != 2) {
		fprintf(stderr, "Invalid image size (error loading '%s')\n", filename);
		exit(1);
	}

	if (fscanf(fp, "%d", &rgb_comp_color) != 1) {
		fprintf(stderr, "Invalid rgb component (error loading '%s')\n",
				filename);
		exit(1);
	}

	if (rgb_comp_color != RGB_COMPONENT_COLOR) {
		fprintf(stderr, "'%s' does not have 8-bits components\n", filename);
		exit(1);
	}

	while (fgetc(fp) != '\n')
		;
	img->data = (PPMPixel*) malloc(img->x * img->y * sizeof(PPMPixel));

	if (!img) {
		fprintf(stderr, "Unable to allocate memory\n");
		exit(1);
	}

	if (fread(img->data, 3 * img->x, img->y, fp) != img->y) {
		fprintf(stderr, "Error loading image '%s'\n", filename);
		exit(1);
	}

	fclose(fp);
	return img;
}

__global__ void cudaHistogram (PPMPixel *data, int size, int *h) {

	int i = threadIdx.x + blockIdx.x * blockDim.x,
		stride = blockDim.x * gridDim.x; /* Gives the number of threads in a grid */ 

	while (i < size) {

		/* Implicit conversion from float to int gives the same result of floor() function */
		int r = ( (float) (data[i].red * 4) / 256),
		    g = ( (float) (data[i].green * 4) / 256),
		    b = ( (float) (data[i].blue * 4) / 256);

		int x = r * 16 + g * 4 + b;

		atomicAdd(&h[x], 1);

		i += stride;
	}

}

int main(int argc, char *argv[]) {

	if( argc != 2 ) {
		printf("Too many or no one arguments supplied.\n");
		return 0;
	}

	int i, n;
	char *filename = argv[1]; //Recebendo o arquivo!;
	double start, end;
	
	//scanf("%s", filename);
	PPMImage *image = readPPM(filename);
	n = image->x * image->y;
	
        int *h = (int*)malloc(sizeof(int) * 64);

	/* We consider in the execution delay the memory allocation time */
	start = rtclock();

	/* Allocating memory for image data in the device */
	int image_size = n * sizeof(PPMPixel);
	PPMPixel *cuda_image;
	hipMalloc((void**) &cuda_image, image_size);
	hipMemcpy(cuda_image, image->data, image_size, hipMemcpyHostToDevice);

	/* Allocating memory for histogram in the device */
	int *cuda_h;
	hipMalloc((void**) &cuda_h, 64 * sizeof(int));
	hipMemset(cuda_h, 0, 64 * sizeof(int));

	/* Computes how many blocks will be used. */
	int cuda_blocks = ceil ( (float) n / THREAD_PER_BLOCK );

	cudaHistogram <<< cuda_blocks, THREAD_PER_BLOCK >>> (cuda_image, n, cuda_h);

	/* Copying computed result from device memory */
	hipMemcpy(h, cuda_h, sizeof(int) * 64, hipMemcpyDeviceToHost);

	/* As cudaMemcpy is a blocking call, we do not need to call cudaThreadSynchronize() */
	end = rtclock();

	for (i = 0; i < 64; i++){
		printf("%0.3f ", (float) h[i] / n);
	}
	printf("\n");

	printf("\n%0.6lfs\n", end - start);
	
	/* Cleaning everything up */
	free(h);
	free(image->data);
	free(image);

	hipFree(cuda_image);
	hipFree(cuda_h);
}

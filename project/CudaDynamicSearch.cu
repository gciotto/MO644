#include "hip/hip_runtime.h"
/*
 * CudaDynamicSearch.cpp
 *
 *  Created on: May 14, 2017
 *      Author: gciotto
 */

#include "CudaDynamicSearch.h"
#include "RingElement.h"
#include <math.h>
#include <iostream>
#include <fstream>


typedef struct {

        unsigned int type;
        double length, focal_distance, sextupole_strength;

} ring_element_t;

__global__ void summaryKernel(pos_t *r, double* s) {

        int j = blockDim.x * blockIdx.x + threadIdx.x,
            i = blockDim.y * blockIdx.y + threadIdx.y;

        if (i < N_POINTS_X && j < N_POINTS_Y) {

                int index = i * N_POINTS_X + j;

                double *t = r[index], sum = t[0] + t[1] + t[2] + t[3] + t[4] + t[5];

                if (!isfinite(sum))
                        s[index] = 1.0;
                else {
                        s[index] = 0.0;
                        for (unsigned int k = 0; k < 6; k++)
                                s[index] += t[k] * t[k];

                        s[index] = sqrt(s[index]);
                }
        }

}

__global__ void dynamicSearchKernel(ring_element_t* c, pos_t *d, unsigned int turns, unsigned int repeat, unsigned int size) {

        double x[2] = {-0.10, +0.10};
        double y[2] = {+0.000, +0.003};

        int j = blockDim.x * blockIdx.x + threadIdx.x,
            i = blockDim.y * blockIdx.y + threadIdx.y;

        if (i < N_POINTS_Y && j < N_POINTS_X) {

		int index = i * N_POINTS_X + j;

#ifdef CUDA_INTRINSICS
                double posx = __dadd_rn(x[0], __dmul_rn(i, __ddiv_rn(x[1] - x[0], N_POINTS_X - 1.0))),
                       posy = __dadd_rn(y[0], __dmul_rn(j, __ddiv_rn(y[1] - y[0], N_POINTS_Y - 1.0)));
#elif CUDA_FMA
		double  posx = __fma_rn(i, (x[1] - x[0]) * __drcp_rn(N_POINTS_X - 1.0), x[0]),
			posy = __fma_rn(j, (y[1] - y[0]) * __drcp_rn(N_POINTS_Y - 1.0), y[0]);
#else
                double posx = x[0] + i*(x[1] - x[0])/(N_POINTS_X - 1.0),
                       posy = y[0] + j*(y[1] - y[0])/(N_POINTS_Y - 1.0);
#endif

                pos_t r = {posx, 0, posy, 0, 0, 0};

                for (unsigned int k = 0; k < turns; k++) {
                        for (unsigned int l = 0; l < repeat; l++) 
                                for (unsigned int m = 0; m < size; m++) {

                                        ring_element_t aux = c[m];

                                        if (aux.type == RingElement::DRIFT) {

						#ifdef CUDA_INTRINSICS
						r[0] = __dadd_rn(r[0], __dmul_rn(aux.length, r[1]));
						r[2] = __dadd_rn(r[2], __dmul_rn(aux.length, r[3]));
						#elif CUDA_FMA
						r[0] = __fma_rn(aux.length, r[1], r[0]);
						r[2] = __fma_rn(aux.length, r[3], r[2]);
						#else
						r[0] += aux.length * r[1];
						r[2] += aux.length * r[3];
						#endif
                                        }
                                        else if (aux.type == RingElement::QUADRUPOLE) {

						#ifdef CUDA_INTRINSICS
						r[1] = __dadd_rn(r[1], __dmul_rn(-1.0, __ddiv_rn(r[0], aux.focal_distance)));
						r[3] = __dadd_rn(r[3], __ddiv_rn(r[2], aux.focal_distance));
						#elif CUDA_FMA
						r[1] = __fma_rn(-r[0], __drcp_rn(aux.focal_distance), r[1]);
						r[3] = __fma_rn( r[2], __drcp_rn(aux.focal_distance), r[3]);
						#else
						r[1] += -r[0]/aux.focal_distance;
						r[3] += r[2]/aux.focal_distance;
						#endif
                                        }
                                        else if (aux.type == RingElement::SEXTUPOLE) {

						#ifdef CUDA_INTRINSICS
						r[1] = __dadd_rn(r[1], __dmul_rn(__dmul_rn(aux.sextupole_strength, aux.length), __dadd_rn(__dmul_rn(r[0],r[0]), __dmul_rn(-1.0, __dmul_rn(r[2], r[2])))));
						r[3] = __dadd_rn(r[3], __dmul_rn(__dmul_rn(aux.sextupole_strength, aux.length), __dmul_rn(2.0, __dmul_rn(r[0], r[2]))));
						#elif CUDA_FMA
						r[1] = __fma_rn(aux.sextupole_strength * aux.length, (r[0]*r[0] - r[2]*r[2]), r[1]);
						r[3] = __fma_rn(aux.sextupole_strength * aux.length, 2.0 * r[0] * r[2], r[3]);
						#else
						r[1] += aux.sextupole_strength * aux.length * (r[0]*r[0] - r[2]*r[2]);
						r[3] += aux.sextupole_strength * aux.length * 2 * r[0]*r[2];
						#endif
                                        }
                                }
		}

                for (unsigned int k = 0; k < 6; k++)
                        d[index][k] = r[k];

        }
}


CudaDynamicSearch::~CudaDynamicSearch() {

    /* Clears everything up */
    this->ring.clear();

    hipFree(this->cuda_result);
}

int CudaDynamicSearch::dynamical_aperture_search() {
      
        ring_element_t *ring_element = (ring_element_t*) malloc (this->ring.size() * sizeof(ring_element_t)),
                       *cuda_ring_element = NULL;

        for (unsigned int i = 0; i < this->ring.size(); i++) {
                ring_element[i].type = this->ring[i]->getType();
                ring_element[i].length = this->ring[i]->getLength();
                ring_element[i].focal_distance = 0;
                ring_element[i].sextupole_strength = 0;

                if (ring_element[i].type == RingElement::QUADRUPOLE)
                        ring_element[i].focal_distance = ((Quadrupole*) this->ring[i])->getFocalDistance();
                else if (ring_element[i].type == RingElement::SEXTUPOLE)
                        ring_element[i].sextupole_strength = ((Sextupole*) this->ring[i])->getSextupoleStrength();
        }

	/* Allocates memory in the device */
	hipMalloc ((void**) &cuda_ring_element, this->ring.size() * sizeof(ring_element_t));
	hipMemcpy(cuda_ring_element, ring_element, this->ring.size() * sizeof(ring_element_t), hipMemcpyHostToDevice);

	/* Computes grid dimension */
	dim3 dimGrid( ceil( (float) N_POINTS_X / CudaDynamicSearch::THREAD_PER_BLOCK ), ceil( (float) N_POINTS_Y / CudaDynamicSearch::THREAD_PER_BLOCK));

	/* Computes block dimensions (X, Y) */
	dim3 dimBlock(CudaDynamicSearch::THREAD_PER_BLOCK, CudaDynamicSearch::THREAD_PER_BLOCK);

	/* Copies result to the host */
	dynamicSearchKernel<<< dimGrid, dimBlock >>>(cuda_ring_element, this->cuda_result, this->turns, this->repeat, this->ring.size());

        hipMemcpy(this->result_set, this->cuda_result, N_POINTS_X * N_POINTS_Y * sizeof(pos_t), hipMemcpyDeviceToHost);

	unsigned int p = 0;
	for (unsigned int i = 0; i < N_POINTS_X ; i++) 
		for (unsigned int j = 0; j < N_POINTS_Y ; j++){
			unsigned int index = i * N_POINTS_X + j;
			if (this->testSolution(this->result_set[index]))
				printf ("%f %f %f %f %f %f (%d / %d) - (%d)\n", this->result_set[index][0], this->result_set[index][1], this->result_set[index][2], this->result_set[index][3], 
                                                                        this->result_set[index][4], this->result_set[index][5], ++p , N_POINTS_X * N_POINTS_Y, index);
		
		}

        free(ring_element);

        hipFree(cuda_ring_element);

	return 0;
}

void CudaDynamicSearch::plot() {

        if (this->result_set != NULL) {

                /* Allocates array for results */
                double *cuda_r, *host_r = (double*) malloc (N_POINTS_X * N_POINTS_Y * sizeof(double));
                std::ofstream out_file;

                hipMalloc((void**) &cuda_r, N_POINTS_X * N_POINTS_Y * sizeof(double));

                /* Computes grid dimension */
                dim3 dimGrid( ceil( (float) N_POINTS_X / CudaDynamicSearch::THREAD_PER_BLOCK ), ceil( (float) N_POINTS_Y / CudaDynamicSearch::THREAD_PER_BLOCK));

                /* Computes block dimensions (X, Y) */
                dim3 dimBlock(CudaDynamicSearch::THREAD_PER_BLOCK, CudaDynamicSearch::THREAD_PER_BLOCK);

        	summaryKernel<<< dimGrid, dimBlock >>>(this->cuda_result, cuda_r);

                hipMemcpy(host_r, cuda_r, N_POINTS_X * N_POINTS_Y * sizeof(double), hipMemcpyDeviceToHost);

                out_file.open ("plot_cudadynamicsearch.dat");
                for (unsigned int i = 0; i < N_POINTS_X; i++) {
                        for (unsigned int j = 0; j < N_POINTS_Y; j++) {

                                int index = i * N_POINTS_X + j;
                                out_file << host_r[index] << " ";
                        }

                        out_file << std::endl;
                }

                out_file.close();

                free(host_r);
                hipFree(cuda_r);
        }
}

